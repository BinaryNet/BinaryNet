
#include <hip/hip_runtime.h>
/*
Copyright 2016 Anonymous researcher(s)

This file is part of BinaryNet.

BinaryNet is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

BinaryNet is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with BinaryNet.  If not, see <http://www.gnu.org/licenses/>.
*/

// CUDA tutorial: http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf
__global__ void gemm(float *a,float *b, float *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if( col < k && row < m) 
    {
        float sum = 0;
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

// 32 single float array ->  32 bits unsigned int
__device__ unsigned int concatenate(float* array)
{
    unsigned int rvalue=0;
    unsigned int sign;
    
    for (int i = 0; i < 32; i++)
    {
        sign = (array[i]>=0);
        rvalue = rvalue | (sign<<i);
    }
    
    return rvalue;
}

// 32 bits unsigned int -> 32 single float array
__device__ float* deconcatenate(unsigned int x)
{
    float * array = new float[32];
    
    for (int i = 0; i < 32; i++)    
    {   
        array[i] = (x & ( 1 << i )) >> i;
    }
    
    return array;
}

__global__ void concatenate_rows_kernel(float *a, unsigned int *b, int size)
{ 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<size) b[i] = concatenate(&a[i*32]);
}

__global__ void concatenate_cols_kernel(float *a, unsigned int *b, int m, int n)
{   

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(j<n){
        float * array = new float[32];
        for(int i=0; i<m; i+=32){
            for(int k=0; k<32;k++) array[k] = a[j + n*(i+k)];
            b[j+n*i/32]=concatenate(array); 
        } 
        delete[] array;
    }
}

__global__ void deconcatenate_rows_kernel(unsigned int *a, float *b, int size)
{ 
    float * array;
    
    for(int i=0; i<size; i+=32)
    {
        array = deconcatenate(a[i/32]);
        for (int k=0;k<32;k++) b[i+k] = array[k];
        delete[] array;
    }
}

__global__ void xnor_gemm(unsigned int *a, unsigned int *b, float *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if( col < k && row < m) 
    {   
        float sum = 0;
        for(int i = 0; i < n; i++) 
        {   
            sum += (float)__popc(~(a[row * n + i]^b[i * k + col]));
        }
        c[row * k + col] = 2*sum-32*n;
    }
}